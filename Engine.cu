#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include "Engine.cuh"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

//#include "Stencil.h"
template<typename dq,typename p>
__global__ void kernel(sogol::Cell<dq::d, dq::q>* c,int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if ((c[i].type != 2)&&(i<size)) {

		p::collide(c[i]);

		c[i].swap();
		
	}
	__syncthreads();
}
template<typename dq, typename p>
__global__ void kernel2(sogol::Cell<dq::d, dq::q>* c, int size, sogol::Vectorxd<dq::d, int> mask) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if ((c[i].type != 2) && (i < size)) {

		p::collide(c[i]);
		
		c[i].swap();
		

	}
	__syncthreads();
	int next;
	int half = (dq::q - 1) / 2;
	auto ck = dq::c;
		if ((c[i].type == 1)&&(i<size)) {
			for (int k = 1; k <= half; k++) {
			    next = i + ck[k] * mask;
				if (next < 0) { continue; }
				if (next >= size) { continue; }
				__syncthreads();



				assert(next >= 0);
				assert(next < size);
				//swap
				float t = c[next][k];
				c[next][k] =c[i].opposite(k);
				c[i].opposite(k) = t;


			}
		}
	
}
template<typename dq>
sogol::Cell<dq::d, dq::q>*  initCu(sogol::Cell<dq::d, dq::q>* cell, int size) {
	std::cout << "init engine";
	int device = -1;
	int bytes = size * sizeof(sogol::Cell<dq::d, dq::q>);
	hipGetDevice(&device);
	sogol::Cell<dq::d, dq::q>* c;
	hipMallocManaged(& c, bytes);
	std::memcpy(c, cell, bytes);
	cell = c;
	hipMemPrefetchAsync(&cell, size * sizeof(sogol::Cell<dq::d, dq::q>), device);
	hipDeviceSynchronize();
	std::cout << "engine ready";
	return cell;

}
template sogol::Cell<2,9>* initCu<sogol::DQ<2,9>>(sogol::Cell<2,9>* c, int size);
 
template<typename dq,typename p>
void runCu(sogol::Cell<dq::d, dq::q>* cell, int size, sogol::Vectorxd<dq::d, int> mask) {
	std::cout << "run engine";
	int device = -1;
	int bytes = size * sizeof(sogol::Cell<dq::d, dq::q>);
	hipGetDevice(&device);
	kernel<dq,p> << <1 + (size / 16), 16 >> > (cell, size);
	hipDeviceSynchronize();
}
template void runCu<sogol::DQ<2,9>,sogol::BGK<sogol::DQ<2,9>>>(sogol::Cell<2, 9>* c, int size, sogol::Vectorxd<2, int> mask);


