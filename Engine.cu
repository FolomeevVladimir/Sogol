#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include "Engine.cuh"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
template<unsigned d, unsigned q>
__global__ void kernel(sogol::Cell<d, q>* c,int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size) { c[i].f.data[0] += 10; }
	__syncthreads();
}
template<unsigned d, unsigned q>
sogol::Cell<d, q>*  initCu(sogol::Cell<d, q>* cell, int size) {
	std::cout << "init engine";
	int device = -1;
	int bytes = size * sizeof(sogol::Cell<d, q>);
	hipGetDevice(&device);
	sogol::Cell<d, q>* c;
	hipMallocManaged(& c, bytes);
	std::memcpy(c, cell, bytes);
	cell = c;
	hipMemPrefetchAsync(&cell, size * sizeof(sogol::Cell<d, q>), device);
	hipDeviceSynchronize();
	std::cout << "engine ready";
	return cell;

}
template sogol::Cell<2, 9>* initCu<2,9>(sogol::Cell<2, 9>* c, int size);
 
template<unsigned d, unsigned q>
void runCu(sogol::Cell<d, q>* cell, int size) {
	std::cout << "run engine";
	int device = -1;
	int bytes = size * sizeof(sogol::Cell<d, q>);
	hipGetDevice(&device);
	kernel<d, q> << <1 + (size / 16), 16 >> > (cell, size);
	hipDeviceSynchronize();
}
template void runCu<2, 9>(sogol::Cell<2, 9>* c, int size);


